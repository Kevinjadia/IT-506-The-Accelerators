#include <iostream>
#include <sys/time.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 32
#define TILE_SIZE 32
#define THRESHOLD 512
using namespace std;

__global__ void split(double *C11, double *C12, double *C21, double *C22, double *C, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i < n && j < n) {
		C11[i * n + j] = C[i * 2 * n + j];
		C12[i * n + j] = C[i * 2 * n + j + n];
		C21[i * n + j] = C[(i + n) * 2 * n + j];
		C22[i * n + j] = C[(i + n) * 2 * n + j + n];
	}
}

__global__ void merge(double *C11, double *C12, double *C21, double *C22, double *C, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i < n && j < n) {
		C[i * 2 * n + j] = C11[i * n + j];
		C[i * 2 * n + j + n] = C12[i * n + j];
		C[(i + n) *2 * n + j] = C21[i * n + j];
		C[(i + n) * 2 * n + j + n] = C22[i * n + j];
	}
}

__global__ void add(double *A, double *B, double *C, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i < n && j < n) {
		C[i * n + j] = A[i * n + j] + B[i * n + j];
	}
}


__global__ void sub(double *A, double *B, double *C, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i < n && j < n) {
		C[i * n + j] = A[i * n + j] - B[i * n + j];
	}
}


__global__ void mul(double *A, double *B, double *C, int n) {
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	if(i < n && j < n) {
		C[i * n + j] = 0;
		for(int k = 0; k < n; k++) {
			C[i * n + j] += A[i * n + k] * B[k * n + j];
		}
	}
}

__global__ void mul_tiled(double* A, double* B, double* C, int width){

    __shared__ double shareA[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double shareB[BLOCK_SIZE][BLOCK_SIZE];
    int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;
    int row = by * BLOCK_SIZE + ty;
    int col = bx * BLOCK_SIZE + tx;
    double temp = 0;
	int no_of_blocks= width/BLOCK_SIZE;

	if(row<width && col<width){
        for(int i = 0; i < no_of_blocks; ++i){
            shareA[ty][tx] = A[row*width + (i*BLOCK_SIZE + tx)];
            shareB[ty][tx] = B[(i*BLOCK_SIZE + ty)*width + col];
            __syncthreads();

            for(int k = 0; k < BLOCK_SIZE; ++k){
                temp += shareA[ty][k] * shareB[k][tx];
            }
            __syncthreads();
        }
        C[row*width + col] = temp;
	}
}

__global__ void mul_add(double *A, double *B, double *T, double *C, int n) {
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	if(i < n && j < n) {
		C[i * n + j] = T[i * n + j];
		for(int k = 0; k < n; k++) {
			C[i * n + j] += A[i * n + k] * B[k * n + j];
		}
	}
}

__global__ void mul_sub_inc(double *A, double *B, double *T, double *C1, double *C2, int n) {
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	if(i < n && j < n) {
		C1[i * n + j] = 0;
		for(int k = 0; k < n; k++) {
			C1[i * n + j] += A[i * n + k] * B[k * n + j];
		}
		C1[i * n + j] = T[i * n + j] - C1[i * n + j]; 
		C2[i * n + j] += T[i * n + j];
	}
}

__global__ void mul_inc_inc_inc(double *A, double *B, double *C, double *T, double *C1, double *C2, int n) {
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	if(i < n && j < n) {
		C[i * n + j] = 0;
		for(int k = 0; k < n; k++) {
			C[i * n + j] += A[i * n + k] * B[k * n + j];
		}
		C1[i * n + j] += C[i * n + j]; 
		C2[i * n + j] += C1[i * n + j];
		C1[i * n + j] += T[i * n + j];
	}
}

//to verify 
void compare(double *res1, double *res2, int n) {
	int fail = 0;
	for(int i = 0; i < n; i++) {
		double a, b;
		a = res1[i] < 0 ? -res1[i] : res1[i];
		b = res2[i] < 0 ? -res2[i] : res2[i];
		if(a < 0.01 && b < 0.01) {
			continue;
		}
		if(i<5) {
			printf("i = %d\t%lf\t%lf\n", i, a, b);
		}
		double diff = (a - b) / (a + 0.000001);
		if(diff < 0) {
			diff= -diff;
		}
		if(diff>0.0005) {
			fail++;
		}
	}
	printf("Number of errors: %d\n", fail);
}

void strassen(double *A, double *B, double *C, int n) {
	

	double *A_gpu, *B_gpu, *C_gpu;
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);

	//Allocating memories to gpu variables
	hipMalloc((void **)&A_gpu, sizeof(double) * n * n);
	hipMalloc((void **)&B_gpu, sizeof(double) * n * n);
	hipMalloc((void **)&C_gpu, sizeof(double) * n * n);


	//copying data from host to device
	hipMemcpy(A_gpu, A, sizeof(double) * n * n, hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, sizeof(double) * n * n, hipMemcpyHostToDevice);

	

	if(n <= 512)
	{
		dim3 grid2((size_t)ceil((double)n / (double)block.x), (size_t)ceil((double)n / (double)block.y));
		mul<<<grid2, block>>>(A_gpu, B_gpu, C_gpu, n);
		hipDeviceSynchronize();
	}else{
		int m = n >> 1;
		dim3 grid((size_t)ceil((double)m / (double)block.x), (size_t)ceil((double)m / (double)block.y));
		double *A11, *A12, *A21, *A22, *B11, *B12, *B21, *B22, *C11, *C12, *C21, *C22, *T1, *T2;
		hipMalloc((void **)&A11, sizeof(double) * m * m);
		hipMalloc((void **)&A12, sizeof(double) * m * m);
		hipMalloc((void **)&A21, sizeof(double) * m * m);
		hipMalloc((void **)&A22, sizeof(double) * m * m);
		hipMalloc((void **)&B11, sizeof(double) * m * m);
		hipMalloc((void **)&B12, sizeof(double) * m * m);
		hipMalloc((void **)&B21, sizeof(double) * m * m);
		hipMalloc((void **)&B22, sizeof(double) * m * m);
		hipMalloc((void **)&C11, sizeof(double) * m * m);
		hipMalloc((void **)&C12, sizeof(double) * m * m);
		hipMalloc((void **)&C21, sizeof(double) * m * m);
		hipMalloc((void **)&C22, sizeof(double) * m * m);
		hipMalloc((void **)&T1, sizeof(double) * m * m);
		hipMalloc((void **)&T2, sizeof(double) * m * m);
				

		split<<<grid, block>>>(A11, A12, A21, A22, A_gpu, m); 
		hipDeviceSynchronize();
		split<<<grid, block>>>(B11, B12, B21, B22, B_gpu, m); 
		hipDeviceSynchronize();		
		sub<<<grid, block>>>(A11, A21, T1, m);
		hipDeviceSynchronize();
		sub<<<grid, block>>>(B22, B12, T2, m);
		hipDeviceSynchronize();
		mul_tiled<<<grid, block>>>(T1, T2, C21, m);
		hipDeviceSynchronize();
		add<<<grid, block>>>(A21, A22, T1, m);
		hipDeviceSynchronize();
		sub<<<grid, block>>>(B12, B11, T2, m);
		hipDeviceSynchronize();
		mul_tiled<<<grid, block>>>(T1, T2, C22, m);
		hipDeviceSynchronize();
		sub<<<grid, block>>>(T1, A11, T1, m);
		hipDeviceSynchronize();
		sub<<<grid, block>>>(B22, T2, T2, m);
		hipDeviceSynchronize();
		mul_tiled<<<grid, block>>>(T1, T2, C11, m);
		hipDeviceSynchronize();
		sub<<<grid, block>>>(A12, T1, T1, m);
		hipDeviceSynchronize();
		mul_add<<<grid, block>>>(T1, B22, C22, C12, m);
		hipDeviceSynchronize();
		mul_inc_inc_inc<<<grid, block>>>(A11, B11, T1, C21, C11, C12, m);
		hipDeviceSynchronize();
		sub<<<grid, block>>>(T2, B21, T2, m);
		hipDeviceSynchronize();
		mul_sub_inc<<<grid, block>>>(A22, T2, C11, C21, C22, m);
		hipDeviceSynchronize();
		mul_add<<<grid, block>>>(A12, B21, T1, C11, m);
		hipDeviceSynchronize();

		merge<<<grid, block>>>(C11, C12, C21, C22, C_gpu, m);	
		hipDeviceSynchronize();
			
			
		hipFree(A11); 
		hipFree(A12); 
		hipFree(A21); 
		hipFree(A22); 
		hipFree(B11); 
		hipFree(B12); 
		hipFree(B21); 
		hipFree(B22); 
		hipFree(T1);
		hipFree(T2);	
	}

	hipMemcpy(C, C_gpu, sizeof(double) * n * n, hipMemcpyDeviceToHost);

    hipFree(A_gpu);
	hipFree(B_gpu);
	hipFree(C_gpu);
}

void serial_mm(double *hostA,double *hostB,double *C_cmp,int n){
	for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            C_cmp[i*n+j] = 0;
 
            for (int k = 0; k < n; k++) {
                C_cmp[i*n+j] += hostA[i*n+k] * hostB[k*n+j];
            }   
        }
    }
}




int main() 
{

	double *hostA,*hostB,*hostC,*C_cmp; 
    
	int minSize=pow(2,1);
    int maxSize=pow(2,13);
    int size,k=1;
	clock_t s1,e1;
	double walltime;

	hipEvent_t start, stop;
	float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);



	for(size=minSize;size<=maxSize;size*=2,k++){

        int tot_allocation_size = size * size * sizeof(double);
          //Dynamically Allocating Memory
		
        hostA = (double*)malloc(tot_allocation_size); 
        hostB = (double*)malloc(tot_allocation_size); 
        hostC = (double*)malloc(tot_allocation_size); 
		C_cmp = (double*)malloc(tot_allocation_size); 


		  // initialization of A B by some random variables
		for(int i = 0; i < size; i++) {
			for(int j = 0; j < size; j++) {
				hostA[i * size + j] = 2.00;
				hostB[i * size + j] = 2.00;
				hostC[i * size + j] = 0;
				C_cmp[i * size + j] = 0;
			}	
		}
		
		hipEventRecord(start);

			strassen(hostA, hostB, hostC, size);
			
		hipEventRecord(stop);

		hipEventSynchronize(stop);

		hipEventElapsedTime(&milliseconds, start, stop);

		s1=clock();
		//serial_mm(hostA, hostB, C_cmp, size);
		e1=clock();
		walltime=(e1-s1)/(double)CLOCKS_PER_SEC;
		printf("\n -------------------------------------\n");
		//compare(hostC, C_cmp,size * size);
    	
    	printf("\n input size:%d x %d(%d) \t Serial time:%lf \t Parallel time:%lf\n",size,size,k,walltime,milliseconds /1000);
		

	}
	
	return 0;
}